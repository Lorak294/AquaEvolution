#include "hip/hip_runtime.h"
#include "headers/simulation_kernels.cuh"
#include "headers/helper_math.cuh"
#include <hip/hip_runtime_api.h>

#ifndef __HIPCC__  
#define __HIPCC__
#endif
#include <hip/device_functions.h>

// ---------------------------------------------------------------- BUCKET SORT FUNCTIONS -----------------------------------------------------------------------

__global__ void calculateAlgaeCellPositions(AquariumSoA aquarium, s_scene scene)
{
	extern __shared__ uint32_t count[];
	count[FISH_COUNT_ID] = *aquarium.fishes.count;
	count[ALGAE_COUNT_ID] = *aquarium.algae.count;
	__syncthreads();

	const uint32_t start_val = blockIdx.x * blockDim.x + threadIdx.x;
	const uint32_t incr_val = blockDim.x * gridDim.x;

	for (int id = start_val;
		id < scene.cellX*scene.cellY;
		id += incr_val)
	{
		scene.cellBucketSizes[id] = 0;
		for (uint64_t i = 0; i < count[ALGAE_COUNT_ID]; i++)
		{
			uint algaCellId = (uint)(aquarium.algae.positions.x[i] / scene.cellWidth) + (uint)(aquarium.algae.positions.y[i] / scene.cellHieght) * scene.cellX;
			if (id == algaCellId)
			{
				scene.cellArray[calc2DIdx<int>(id, scene.cellBucketSizes[id], scene.pitch)] = i;
				scene.cellBucketSizes[id]++;
				//printf("alga[%u] pos:[%f,%f]\t - cellXid [%u]\t - cellYid [%u] -> cellID: [%u]\n", i, aquarium.algae.positions.x[i], aquarium.algae.positions.y[i], (uint)(aquarium.algae.positions.x[i] / scene.cellWidth), (uint)(aquarium.algae.positions.y[i] / scene.cellHieght), algaCellId);
			}
		}
		//printf("cell [%d] has [%d] algae\n", id, scene.cellBucketSizes[id]);
	}
}
template<typename T>
__device__ int calc2DIdx(int row, int col, size_t pitch)
{
	return row*pitch / sizeof(T) + col;
}

// ---------------------------------------------------------------- FINDING CLOSEST ALGA FUNCTIONS -----------------------------------------------------------------------

__device__ int findClosestAlga(AquariumSoA* aquarium, s_scene* scene, uint32_t fishId, float* distToBeat)
{
	uint2 fishCell = { (uint)(aquarium->fishes.positions.x[fishId] / scene->cellWidth), (uint)(aquarium->fishes.positions.y[fishId] / scene->cellHieght) };

	//check same cell
	int closest_algae_id = findClosestAlgaInCell(aquarium,scene,fishId,fishCell, distToBeat);
	
	// check cells above
	if (fishCell.y > 0)
	{
		// directly above
		closest_algae_id = findClosestAlgaInCell(aquarium, scene, fishId, {fishCell.x, fishCell.y -1}, distToBeat);
		// left above
		if (fishCell.x > 0)
		{
			closest_algae_id = findClosestAlgaInCell(aquarium, scene, fishId, { fishCell.x - 1, fishCell.y - 1 }, distToBeat);
		}
		// right above
		if (fishCell.x < scene->cellX - 1)
		{
			closest_algae_id = findClosestAlgaInCell(aquarium, scene, fishId, { fishCell.x + 1, fishCell.y - 1 }, distToBeat);
		}
	}
	// check cell below
	if (fishCell.y < scene->cellY - 1)
	{
		// directly below
		closest_algae_id = findClosestAlgaInCell(aquarium, scene, fishId, { fishCell.x, fishCell.y + 1 }, distToBeat);
		// left below
		if (fishCell.x > 0)
		{
			closest_algae_id = findClosestAlgaInCell(aquarium, scene, fishId, { fishCell.x - 1, fishCell.y + 1 }, distToBeat);
		}
		// right below
		if (fishCell.x < scene->cellX - 1)
		{
			closest_algae_id = findClosestAlgaInCell(aquarium, scene, fishId, { fishCell.x + 1, fishCell.y + 1 }, distToBeat);
		}
	}
	// check cell on the left
	if (fishCell.x > 0)
	{
		closest_algae_id = findClosestAlgaInCell(aquarium, scene, fishId, { fishCell.x - 1, fishCell.y }, distToBeat);
	}
	// check cell on the right
	if (fishCell.x < scene->cellX - 1)
	{
		closest_algae_id = findClosestAlgaInCell(aquarium, scene, fishId, { fishCell.x + 1, fishCell.y }, distToBeat);
	}

	return closest_algae_id;
}

__device__ int findClosestAlgaInCell(AquariumSoA* aquarium, s_scene* scene, uint32_t fishId, uint2 cell, float* distToBeat)
{
	int closest_alga_id = -1;
	int cellArrayIdx = cell.x + cell.y * scene->cellX;

	for (int i = 0; i < scene->cellBucketSizes[cellArrayIdx]; i++)
	{
		int alga_id = scene->cellArray[calc2DIdx<int>(cellArrayIdx, i, scene->pitch)];
		float curr_dist = algae_in_sight_dist(aquarium, fishId, alga_id);

		if (curr_dist != -1 && curr_dist < *distToBeat)
		{
			closest_alga_id = alga_id;
			*distToBeat = curr_dist;
		}
	}
	return closest_alga_id;
}

// returns algae distance or -1 if fish cannot see the algae
__device__ float algae_in_sight_dist(AquariumSoA* aquarium, uint32_t fishId, size_t algaId)
{
	float2 algaPos = { aquarium->algae.positions.x[algaId], aquarium->algae.positions.y[algaId] };
	float2 fishPos = { aquarium->fishes.positions.x[fishId], aquarium->fishes.positions.y[fishId] };
	float2 fishVec = { aquarium->fishes.directionVecs.x[fishId], aquarium->fishes.directionVecs.y[fishId] };

	float2 vecToAlga = algaPos - fishPos;

	float dist = length(vecToAlga);

	//check distance
	if (dist > aquarium->fishes.stats.sightDist[fishId])
		return -1.f;

	// check angle
	float cosine = dot(fishVec, vecToAlga/dist);
	if (cosine < aquarium->fishes.stats.sightAngle[fishId])
		return -1.f;

	return dist;
}

// ---------------------------------------------------------------- MAIN SIMULATION FUNCTIONS -----------------------------------------------------------------------

__global__ void simulate_generation(AquariumSoA aquarium, s_scene scene, hiprandState* generators)
{
	extern __shared__ uint32_t count[];
	count[FISH_COUNT_ID] = *aquarium.fishes.count;
	count[ALGAE_COUNT_ID] = *aquarium.algae.count;
	__syncthreads();

	const uint32_t start_val = blockIdx.x * blockDim.x + threadIdx.x;
	const uint32_t incr_val = blockDim.x * gridDim.x;

	for (int j = 0; j < TICKS_PER_GENERATION; ++j)
	{
		fish_decision(&aquarium, &scene, start_val, incr_val);
		__syncthreads();

		algae_decision(&aquarium, start_val, incr_val);
		__syncthreads();

		fish_move(&aquarium, start_val, incr_val);
		__syncthreads();

		algae_move(&aquarium, start_val, incr_val);
		__syncthreads();
	}

	if (start_val != 0) return;
	fish_reproduction(&aquarium, start_val, incr_val, generators);
	algae_reproduction(&aquarium, start_val, incr_val, generators);

}

__device__ void fish_decision(AquariumSoA* aquarium, s_scene* scene, uint32_t start_val, uint32_t incr_val)
{
	uint32_t id;
	extern __shared__ uint32_t count[]; 
	
	// Fish decision
	for (id = start_val;
		id < count[FISH_COUNT_ID];
		id += incr_val)
	{
		if (aquarium->fishes.alives[id] == FishAliveEnum::DEAD) continue;

		float fish_x = aquarium->fishes.positions.x[id];
		float fish_y = aquarium->fishes.positions.y[id];

		// Loop through all algea and find closest one
		int closest_algae_id = -1;
		float closest_algae_dist = FLT_MAX;
		closest_algae_id = findClosestAlga(aquarium, scene, id, &closest_algae_dist);

		if (closest_algae_id == -1)
		{
			aquarium->fishes.nextDecisions[id] = FishDecisionEnum::NONE;
			return;
		}

		//printf("FISH [%u] FOUND TARGET --- ALGAE[%u]\n", id, closest_algae_id);

		// update fishes vector if there is any algea on map
		float algae_x = aquarium->algae.positions.x[closest_algae_id];
		float& vec_x = aquarium->fishes.directionVecs.x[id];
		vec_x = algae_x - fish_x;

		float algae_y = aquarium->algae.positions.y[closest_algae_id];
		float& vec_y = aquarium->fishes.directionVecs.y[id];
		vec_y = algae_y - fish_y;

		float d = sqrtf(vec_x * vec_x + vec_y * vec_y);
		if (d > 0.01f)
		{
			vec_x /= d;
			vec_y /= d;
		}

		// Check if eating is available
		bool eat_available = closest_algae_dist < 0.01;
		if (eat_available)
		{
			aquarium->fishes.nextDecisions[id] = FishDecisionEnum::EAT;
			aquarium->fishes.interactionEntityIds[id] = closest_algae_id;
			aquarium->algae.alives[closest_algae_id] = false;
		}
		else
		{
			aquarium->fishes.nextDecisions[id] = FishDecisionEnum::MOVE;
		}
	}
}

void algae_decision(AquariumSoA* aquarium, uint32_t start_val, uint32_t incr_val)
{
	uint32_t id;
	extern __shared__ uint32_t count[]; 

	// Algae decision
	for (id = start_val;
		id < count[ALGAE_COUNT_ID];
		id += incr_val)
	{
		if (!aquarium->algae.alives[id]) continue;
		float vec_x = aquarium->algae.directionVecs.x[id];
		float vec_y = aquarium->algae.directionVecs.y[id];
		float denom = sqrtf(vec_x * vec_x + vec_y * vec_y);
		if (denom >= 0.00001f)
		{
			vec_x /= denom;
			vec_y /= denom;
		}

		// Bounces
		// TODO(kutakw): wtf with those boundries
		float pos_x = aquarium->algae.positions.x[id];
		float new_pos_x = pos_x + vec_x * ALGAE_VELOCITY;
		if (new_pos_x < AQUARIUM_LEFT_BORDER || new_pos_x >= AQUARIUM_RIGHT_BORDER)
			vec_x *= -1.0f;

		aquarium->algae.directionVecs.x[id] = vec_x;

		float pos_y = aquarium->algae.positions.y[id];
		float new_pos_y = pos_y + vec_y * ALGAE_VELOCITY;
		if (new_pos_y < AQUARIUM_BOTTOM_BORDER || new_pos_y >= AQUARIUM_TOP_BORDER)
			vec_y *= -1.0f;

		aquarium->algae.directionVecs.y[id] = vec_y;
	}
}

void fish_move(AquariumSoA* aquarium, uint32_t start_val, uint32_t incr_val)
{
	uint32_t id;
	extern __shared__ uint32_t count[]; 

	// Fish move
	for (id = start_val;
		id < count[FISH_COUNT_ID];
		id += incr_val)
	{
		if (aquarium->fishes.alives[id] == FishAliveEnum::DEAD) continue;

		FishDecisionEnum decision = aquarium->fishes.nextDecisions[id];
		float energy = aquarium->fishes.currentEnergy[id];
		switch (decision)
		{
		case FishDecisionEnum::NONE:
			break;
		case FishDecisionEnum::MOVE:
		{
			// Update FISHES pos 
			float& pos_x = aquarium->fishes.positions.x[id];
			float vec_x = aquarium->fishes.directionVecs.x[id];
			pos_x += vec_x * FISH_VELOCITY;

			float& pos_y = aquarium->fishes.positions.y[id];
			float vec_y = aquarium->fishes.directionVecs.y[id];
			pos_y += vec_y * FISH_VELOCITY;
			break;
		}
		case FishDecisionEnum::EAT:
		{
			energy += ALGAE_ENERGY_VALUE;
			break;
		}
		}

		energy -= aquarium->fishes.stats.energyUsage[id];

		// Check if fish alive
		if (energy <= 0)
		{
			printf("fish[%u] is dead\n", id);
			aquarium->fishes.alives[id] = FishAliveEnum::DEAD;
		}

		aquarium->fishes.currentEnergy[id] = min(energy, aquarium->fishes.stats.maxEnergy[id]);
	}
}

void algae_move(AquariumSoA* aquarium, uint32_t start_val, uint32_t incr_val)
{
	uint32_t id;
	extern __shared__ uint32_t count[];

	// ALGAE MOVE
	for (id = start_val;
		id < count[ALGAE_COUNT_ID];
		id += incr_val)
	{
		if (!aquarium->algae.alives[id]) continue;

		float& pos_x = aquarium->algae.positions.x[id];
		float vec_x = aquarium->algae.directionVecs.x[id];
		pos_x += vec_x * ALGAE_VELOCITY;

		float& pos_y = aquarium->algae.positions.y[id];
		float vec_y = aquarium->algae.directionVecs.y[id];
		pos_y += vec_y * ALGAE_VELOCITY;

		float energy = aquarium->algae.currentEnergy[id];
		float algae_height = pos_y / AQUARIUM_TOP_BORDER;
		float energyLoss = aquarium->algae.stats.energyUsage[id];
		float energyGain = lerp(0.000075f, 0.00125f, algae_height);
		energy -= energyLoss;
		energy += energyGain;
		if(energy < 0.0f) 
		{
			aquarium->algae.alives[id] = false;
			printf("algae[%u] is dead\n", id);
			continue;
		}

		energy = fminf(energy, 50.0f);

		aquarium->algae.currentEnergy[id] = energy;
	}
}

void fish_reproduction(AquariumSoA* aquarium, uint32_t start_val, uint32_t incr_val, hiprandState* generators)
{
	uint32_t id;
	extern __shared__ uint32_t count[];
	int fish_count = count[FISH_COUNT_ID];
	hiprandState generator = generators[start_val];

	// Get data
	int new_index = 0;
	for (int i = 0; i < fish_count; ++i)
	{
		if (aquarium->fishes.alives[i] == FishAliveEnum::ALIVE)
		{
			aquarium->fishes.alives[new_index] = aquarium->fishes.alives[i];
			aquarium->fishes.directionVecs.x[new_index] = aquarium->fishes.directionVecs.x[i];
			aquarium->fishes.directionVecs.y[new_index] = aquarium->fishes.directionVecs.y[i];
			aquarium->fishes.currentEnergy[new_index] = aquarium->fishes.currentEnergy[i];
			aquarium->fishes.interactionEntityIds[new_index] = aquarium->fishes.interactionEntityIds[i];
			aquarium->fishes.nextDecisions[new_index] = aquarium->fishes.nextDecisions[i];
			aquarium->fishes.positions.x[new_index] = aquarium->fishes.positions.x[i];
			aquarium->fishes.positions.y[new_index] = aquarium->fishes.positions.y[i];
			aquarium->fishes.stats.size[new_index] = aquarium->fishes.stats.size[i];

			new_index++;
		}
	}

	int new_fish_count = new_index;
	for (int i = 0; i < new_fish_count; ++i)
	{
		if (new_index + 10 >= Aquarium::maxObjCount)
		{
			*aquarium->fishes.count = new_index;
			return;
		}

		float energy = aquarium->fishes.currentEnergy[i];
		if (energy < 35.0f) continue;
		energy -= 10.0f;
		
		int children_count = (((int)hiprand(&generator) & INT_MAX) % 10) + 1;
		for (int j = 0; j < children_count; ++j)
		{
			float offx = hiprand_uniform(&generator) + 1.5f;
			float offy = hiprand_uniform(&generator) + 1.5f;

			aquarium->fishes.alives[new_index] = FishAliveEnum::ALIVE;
			aquarium->fishes.positions.x[new_index] = aquarium->fishes.positions.x[i] + offx;
			aquarium->fishes.positions.y[new_index] = aquarium->fishes.positions.y[i] + offy;
			aquarium->fishes.directionVecs.x[new_index] = 1.0f;
			aquarium->fishes.directionVecs.y[new_index] = 0.0f;
			aquarium->fishes.stats.size[new_index] = aquarium->fishes.stats.size[i] * 1.1f;
			aquarium->fishes.currentEnergy[new_index] = Fish::ENERGY_INITIAL;
			
			new_index++;
		}
		aquarium->fishes.currentEnergy[i] = energy;
	}

	*aquarium->fishes.count = new_index;
	generators[start_val] = generator;
}

void algae_reproduction(
	AquariumSoA* aquarium,
	uint32_t start_val,
	uint32_t incr_val,
	hiprandState* generators
)
{
	uint32_t id;
	extern __shared__ uint32_t count[];
	int algae_count = count[ALGAE_COUNT_ID];
	hiprandState generator = generators[start_val];

	// Get data
	int new_index = 0;
	for (int i = 0; i < algae_count; ++i)
	{
		if (aquarium->algae.alives[i])
		{
			aquarium->algae.positions.x[new_index] = aquarium->algae.positions.x[i];
			aquarium->algae.positions.y[new_index] = aquarium->algae.positions.y[i];
			aquarium->algae.directionVecs.x[new_index] = aquarium->algae.directionVecs.x[i];
			aquarium->algae.directionVecs.y[new_index] = aquarium->algae.directionVecs.y[i];
			aquarium->algae.alives[new_index] = aquarium->algae.alives[i];
			aquarium->algae.currentEnergy[new_index] = aquarium->algae.currentEnergy[i];

			aquarium->algae.stats.size[new_index] = aquarium->algae.stats.size[i];
			aquarium->algae.stats.maxEnergy[new_index] = aquarium->algae.stats.maxEnergy[i];
			aquarium->algae.stats.energyUsage[new_index] = aquarium->algae.stats.energyUsage[i];

			new_index++;
		}
	}

	int new_algae_count = new_index;
	for (int i = 0; i < new_algae_count; ++i)
	{
		if (new_index + 10 >= Aquarium::maxObjCount) break;

		float energy = aquarium->algae.currentEnergy[i];
		if (energy < 28.0f) continue;
		energy -= 5.0f;
		
		int children_count = (((int)hiprand(&generator) & INT_MAX) % 10) + 1;
		for (int j = 0; j < children_count; ++j)
		{
			float offx = 5.0f * hiprand_uniform(&generator) - 2.5f;
			float offy = 5.0f * hiprand_uniform(&generator) - 2.5f;
			float2 vec = { hiprand_uniform(&generator), hiprand_uniform(&generator) };
			vec = normalize(vec);
			float2 pos = { 
				clamp(aquarium->algae.positions.x[i] + offx, AQUARIUM_LEFT_BORDER + 0.1f, AQUARIUM_RIGHT_BORDER - 0.1f), 
				clamp(aquarium->algae.positions.y[i] + offy, AQUARIUM_BOTTOM_BORDER + 0.1f, AQUARIUM_TOP_BORDER - 0.1f), 
			};

			aquarium->algae.alives[new_index] = true;
			aquarium->algae.positions.x[new_index] = pos.x;
			aquarium->algae.positions.y[new_index] = pos.y;
			aquarium->algae.directionVecs.x[new_index] = aquarium->algae.directionVecs.x[i];
			aquarium->algae.directionVecs.y[new_index] = aquarium->algae.directionVecs.y[i];
			aquarium->algae.currentEnergy[new_index] = Algae::initialEnergy;

			aquarium->algae.stats.size[new_index] = aquarium->algae.stats.size[i];
			aquarium->algae.stats.maxEnergy[new_index] = aquarium->algae.stats.maxEnergy[i];
			aquarium->algae.stats.energyUsage[new_index] = aquarium->algae.stats.energyUsage[i];
			
			new_index++;
		}
		aquarium->algae.currentEnergy[i] = energy;
	}

	*aquarium->algae.count = new_index;
	generators[start_val] = generator;
}
