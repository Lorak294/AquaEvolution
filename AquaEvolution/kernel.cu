#include "hip/hip_runtime.h"
#include ""

#include "headers/Shader.h"
#include "headers/rendering.cuh"
#include "headers/aquarium.cuh"

#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdlib>
#include <ctime>
#include <random>

#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>

#define checkCudaErrors(call)                                 \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)


void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void processInput(GLFWwindow *window);
GLFWwindow* createGLWindow();
void createBuffers();
void createBackgroundBuffers();
void createBackgroundBuffers();
void createAlgaBuffers();
void cleanup();
void renderLoop(GLFWwindow* window, Shader shader);
//void copy_bitmap_to_host();
void makeNewGeneration();
void resetAquariumStruct(int algaeCount);
void copyAquariumStructToDevice();
void copyAquariumStructFromDevice();
void freeHostAquariumStruct();
void freeDeviceAquariumStruct();
void mallocHostAquariumStruct(int algaeCount);
void mallocDeviceAquariumStruct(int algaeCount);

// openGL parameters
unsigned int SCR_WIDTH = 1000;
unsigned int SCR_HEIGHT = 1000;
const glm::vec3 SURFACE = { 0.0f, 0.9f, 0.9f };
const glm::vec3 DEEPWATER = { 0.0f, 0.0f, 0.0f };
const glm::vec3 ALGAECOLOR = { 0.0f, 1.0f, 0.0f };

// settings
const float alga::initaialSize = 0.5f;

const unsigned int TX = 16;
const unsigned int TY = 16;

const bool GPU_RNEDER = true;

// global variables 
aquarium hostAquarium;
s_aquarium hostAquariumStruct;
s_aquarium deviceAquariumStruct;
int algaeCount = 1;

double oneFrameTime;
double copyTime;
double renderTime;

unsigned int VBO_bg, VAO_bg, EBO_bg;
unsigned int VBO_alga, VAO_alga, EBO_alga;

int main()
{
	srand(static_cast<unsigned> (time(0)));

	GLFWwindow* window = createGLWindow();

	// create shader
	Shader shader("texture.vs", "texture.fs");

	// creating openGL buffers and drawing
	createBuffers();
	renderLoop(window, shader);

	// cleanup
	cleanup();
	return 0;
}

// -------------------------------------------- OPENGL FUNCTIONS -----------------------------------------------
// process all input: query GLFW whether relevant keys are pressed/released this frame and react accordingly
void processInput(GLFWwindow *window)
{
	if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
	{
		glfwSetWindowShouldClose(window, true);
	}
}
// glfw: whenever the window size changed (by OS or user resize) this callback function executes
void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
	// make sure the viewport matches the new window dimensions; note that width and 
	// height will be significantly larger than specified on retina displays.
	glViewport(0, 0, width, height);

	// update parameters
	SCR_WIDTH = width;
	SCR_HEIGHT = height;
}
GLFWwindow* createGLWindow()
{
	// glfw: initialize and configure
	// ------------------------------
	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

#ifdef __APPLE__
	glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif

	// glfw window creation
	// --------------------
	GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "AquaEvolution", NULL, NULL);
	if (window == NULL)
	{
		std::cout << "Failed to create GLFW window" << std::endl;
		glfwTerminate();
		exit(-1);
	}
	glfwMakeContextCurrent(window);
	glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
	//glfwSetKeyCallback(window, key_callback);

	// glad: load all OpenGL function pointers
	// ---------------------------------------
	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
	{
		std::cout << "Failed to initialize GLAD" << std::endl;
		exit(-1);
	}
	return window;
}
void createBuffers()
{
	createBackgroundBuffers();
	createAlgaBuffers();
}
void createBackgroundBuffers()
{
	// set up vertex data (and buffer(s)) and configure vertex attributes
	float vertices[] =
	{	// coords			// colors 
		 1.0f,  1.0f, 1.0f, SURFACE.r,		SURFACE.g,		SURFACE.b ,		// top right
		 1.0f, -1.0f, 0.0f, DEEPWATER.r,	DEEPWATER.g,	DEEPWATER.b ,	// bottom right
		-1.0f, -1.0f, 0.0f, DEEPWATER.r,	DEEPWATER.g,	DEEPWATER.b ,	// bottom left
		-1.0f,  1.0f, 0.0f, SURFACE.r,		SURFACE.g,		SURFACE.b ,		// top left 
	};
	unsigned int indices[] =
	{
		0, 1, 3,   // first triangle
		1, 2, 3    // second triangle
	};


	glGenVertexArrays(1, &VAO_bg);
	glGenBuffers(1, &VBO_bg);
	glGenBuffers(1, &EBO_bg);

	glBindVertexArray(VAO_bg);

	glBindBuffer(GL_ARRAY_BUFFER, VBO_bg);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO_bg);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

	// position attribute
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);
	// color attribute
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)(3 * sizeof(float)));
	glEnableVertexAttribArray(1);
}
void createAlgaBuffers()
{
	// set up vertex data (and buffer(s)) and configure vertex attributes
	float vertices[] =
	{	// coords			// colors 
		 0.0f,  1.0f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// top 
		 0.7f,  0.7f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// top right
		 1.0f,  0.0f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// left
		 0.7f, -0.7f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// bottom right 
		 0.0f, -1.0f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// bottom
		-0.7f, -0.7f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// bottom left 
		-1.0f,  0.0f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// left 
		-0.7f,  0.7f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// top left 
	};
	unsigned int indices[] =
	{
		4, 5, 6,   
		4, 6, 7,   
		4, 7, 0,    
		4, 0, 1,    
		4, 1, 2,
		4, 2, 3
	};


	glGenVertexArrays(1, &VAO_alga);
	glGenBuffers(1, &VBO_alga);
	glGenBuffers(1, &EBO_alga);

	glBindVertexArray(VAO_alga);

	glBindBuffer(GL_ARRAY_BUFFER, VBO_alga);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO_alga);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

	// position attribute
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);
	// color attribute
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)(3 * sizeof(float)));
	glEnableVertexAttribArray(1);
}

// --------------------------------------------- MEMORY MANAGEMENT FUNCTIONS ------------------------------------
void cleanup()
{
	// de-allocate all resources once they've outlived their purpose:
	glDeleteVertexArrays(1, &VAO_bg);
	glDeleteBuffers(1, &VBO_bg);
	glDeleteBuffers(1, &EBO_bg);

	glDeleteVertexArrays(1, &VAO_alga);
	glDeleteBuffers(1, &VBO_alga);
	glDeleteBuffers(1, &EBO_alga);

	// glfw: terminate, clearing all previously allocated GLFW resources.
	glfwTerminate();
}
// --------------------------------------------- RENDERING FUNCTIONS ----------------------------------------------
void renderLoop(GLFWwindow* window, Shader shader)
{
	// we operate in simple 2D space so form MVP matrices M is enough

	while (!glfwWindowShouldClose(window))
	{
		// allocate aquarium resources

		// input
		processInput(window);

		// render
		glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		//if (GPU_RNEDER)
		//{
		//	dim3 blocks(SCR_WIDTH / TX + 1, SCR_HEIGHT / TY + 1);
		//	dim3 threads(TX, TY);

		//	//render_GPU << <blocks, threads >> > (deviceBitmap, SCR_WIDTH, SCR_HEIGHT);
		//	checkCudaErrors(hipGetLastError());
		//	checkCudaErrors(hipDeviceSynchronize());
		//}
		//else
		//{
		//	render_CPU(hostBitmap, SCR_WIDTH, SCR_HEIGHT);
		//}
		
		// model matrix
		glm::mat4 model = glm::mat4(1.0f);
		shader.use();

		// render background
		glBindVertexArray(VAO_bg);
		shader.setMat4("model", model);
		glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);

		// render algae
		glBindVertexArray(VAO_alga);
		model = glm::scale(model, glm::vec3(0.2, 0.2, 0.2));
		shader.setMat4("model", model);
		glDrawElements(GL_TRIANGLES, 18, GL_UNSIGNED_INT, 0);

		// glfw: swap buffers and poll IO events (keys pressed/released, mouse moved etc.)
		glfwSwapBuffers(window);
		glfwPollEvents();

	}
}

// --------------------------------------------- AQUARIUM MANAGEMENT FUNCTIONS ----------------------------------------------
void makeNewGeneration()
{
	// copy alive algae to new aquarium
	hostAquarium.readFromDeviceStruct(hostAquariumStruct, false);

	// make new algae from each one
	hostAquarium.newGeneration();

	// apply mutations and store algae in new arrays
	resetAquariumStruct(hostAquarium.algae.size());
	hostAquarium.writeToDeviceStruct(hostAquariumStruct);
}

// --------------------------------------------- MEMORY MANAGEMENT FUNCTIONS ----------------------------------------------
void resetAquariumStruct(int algaeCount)
{
	freeHostAquariumStruct();
	mallocHostAquariumStruct(algaeCount);
}
void copyAquariumStructToDevice()
{
	int n = *(hostAquariumStruct.algaeCount);
	checkCudaErrors(hipMemcpy(deviceAquariumStruct.algaeCount, hostAquariumStruct.algaeCount, sizeof(int), hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(deviceAquariumStruct.algae.positions.x, hostAquariumStruct.algae.positions.x, sizeof(float)*n, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(deviceAquariumStruct.algae.positions.y, hostAquariumStruct.algae.positions.y, sizeof(float)*n, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(deviceAquariumStruct.algae.driftingVecs.x, hostAquariumStruct.algae.driftingVecs.x, sizeof(float)*n, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(deviceAquariumStruct.algae.driftingVecs.y, hostAquariumStruct.algae.driftingVecs.y, sizeof(float)*n, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(deviceAquariumStruct.algae.alives, hostAquariumStruct.algae.alives, sizeof(bool)*n, hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMemcpy(deviceAquariumStruct.algae.sizes, hostAquariumStruct.algae.sizes, sizeof(float)*n, hipMemcpyHostToDevice));
}
void copyAquariumStructFromDevice()
{
	// NOTE: auuming sizes of arrays have not changed during kernel execution
	int n = *(hostAquariumStruct.algaeCount);
	
	checkCudaErrors(hipMemcpy(hostAquariumStruct.algaeCount, deviceAquariumStruct.algaeCount, sizeof(int), hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(hostAquariumStruct.algae.positions.x, deviceAquariumStruct.algae.positions.x, sizeof(float)*n, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(hostAquariumStruct.algae.positions.y, deviceAquariumStruct.algae.positions.y, sizeof(float)*n, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(hostAquariumStruct.algae.driftingVecs.x, deviceAquariumStruct.algae.driftingVecs.x, sizeof(float)*n, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(hostAquariumStruct.algae.driftingVecs.y, deviceAquariumStruct.algae.driftingVecs.y, sizeof(float)*n, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(hostAquariumStruct.algae.alives, deviceAquariumStruct.algae.alives, sizeof(bool)*n, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(hostAquariumStruct.algae.sizes, deviceAquariumStruct.algae.sizes, sizeof(float)*n, hipMemcpyHostToDevice));
}
void freeDeviceAquariumStruct()
{
	checkCudaErrors(hipFree(deviceAquariumStruct.algaeCount));

	checkCudaErrors(hipFree(deviceAquariumStruct.algae.positions.x));
	checkCudaErrors(hipFree(deviceAquariumStruct.algae.positions.y));

	checkCudaErrors(hipFree(deviceAquariumStruct.algae.driftingVecs.x));
	checkCudaErrors(hipFree(deviceAquariumStruct.algae.driftingVecs.y));

	checkCudaErrors(hipFree(deviceAquariumStruct.algae.alives));

	checkCudaErrors(hipFree(deviceAquariumStruct.algae.sizes));
}
void freeHostAquariumStruct()
{
	free(hostAquariumStruct.algaeCount);

	free(hostAquariumStruct.algae.positions.x);
	free(hostAquariumStruct.algae.positions.y);

	free(hostAquariumStruct.algae.driftingVecs.x);
	free(hostAquariumStruct.algae.driftingVecs.y);

	free(hostAquariumStruct.algae.alives);

	free(hostAquariumStruct.algae.sizes);
}
void mallocHostAquariumStruct(int algaeCount)
{
	hostAquariumStruct.algaeCount = (int*)malloc(sizeof(int)); // -> may be unnecessary
	*(hostAquariumStruct.algaeCount) = algaeCount;
	hostAquariumStruct.algae.positions.x = (float*)malloc(algaeCount * sizeof(float));
	hostAquariumStruct.algae.positions.y = (float*)malloc(algaeCount * sizeof(float));
	hostAquariumStruct.algae.driftingVecs.x = (float*)malloc(algaeCount * sizeof(float));
	hostAquariumStruct.algae.driftingVecs.y = (float*)malloc(algaeCount * sizeof(float));
	hostAquariumStruct.algae.alives = (bool*)malloc(algaeCount * sizeof(bool));
	hostAquariumStruct.algae.sizes = (float*)malloc(algaeCount * sizeof(float));
}
void mallocDeviceAquariumStruct(int algaeCount)
{
	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.algaeCount, sizeof(int)));

	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.algae.positions.x, sizeof(float)*algaeCount));
	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.algae.positions.y, sizeof(float)*algaeCount));

	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.algae.driftingVecs.x, sizeof(float)*algaeCount));
	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.algae.driftingVecs.y, sizeof(float)*algaeCount));

	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.algae.alives, sizeof(float)*algaeCount));

	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.algae.sizes, sizeof(float)*algaeCount));
}