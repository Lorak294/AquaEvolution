#include "hip/hip_runtime.h"
#include ""

#include "headers/Shader.h"
#include "headers/aquarium.cuh"

#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdlib>
#include <ctime>
#include <random>

#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>

#define checkCudaErrors(call)                                 \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)


void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void processInput(GLFWwindow *window);
GLFWwindow* createGLWindow();
void createBuffers();
void createBackgroundBuffers();
void createBackgroundBuffers();
void createAlgaBuffers();
void createFishBuffers();
void cleanup();
void renderLoop(GLFWwindow* window, Shader shader);
void makeNewGeneration();
void resetAquariumStruct(int algaeCount);
void copyAquariumStructToDevice();
void copyAquariumStructFromDevice();
void freeHostAquariumStruct();
void freeDeviceAquariumStruct();
void mallocHostAquariumStruct(int algaeCount);
void mallocDeviceAquariumStruct(int algaeCount);

// openGL parameters
unsigned int SCR_WIDTH = 1000;
unsigned int SCR_HEIGHT = 1000;
const glm::vec3 SURFACE = { 0.0f, 0.9f, 0.9f };
const glm::vec3 DEEPWATER = { 0.0f, 0.0f, 0.0f };
const glm::vec3 ALGAECOLOR = { 0.0f, 1.0f, 0.0f };
const glm::vec3 FISHCOLOR1 = { 0.94f, 0.54f, 0.09f };
const glm::vec3 FISHCOLOR2 = { 0.85f, 0.7f, 0.2f };

// settings
const float Object::initaialSize = 0.5f;
const unsigned int CELLSX = 10;
const unsigned int CELLSY = 10;

// global variables 
Aquarium hostAquarium;
s_aquarium hostAquariumStruct;
s_aquarium deviceAquariumStruct;

unsigned int VBO_bg, VAO_bg, EBO_bg;
unsigned int VBO_alga, VAO_alga, EBO_alga;
unsigned int VBO_fish, VAO_fish, EBO_fish;

int main()
{
	srand(static_cast<unsigned> (time(0)));

	GLFWwindow* window = createGLWindow();

	// create shader
	Shader shader("texture.vs", "texture.fs");

	// creating openGL buffers and drawing
	createBuffers();
	renderLoop(window, shader);

	// cleanup
	cleanup();
	return 0;
}

// -------------------------------------------- OPENGL FUNCTIONS -----------------------------------------------
// process all input: query GLFW whether relevant keys are pressed/released this frame and react accordingly
void processInput(GLFWwindow *window)
{
	if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
	{
		glfwSetWindowShouldClose(window, true);
	}
}
// glfw: whenever the window size changed (by OS or user resize) this callback function executes
void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
	// make sure the viewport matches the new window dimensions; note that width and 
	// height will be significantly larger than specified on retina displays.
	glViewport(0, 0, width, height);

	// update parameters
	SCR_WIDTH = width;
	SCR_HEIGHT = height;
}
GLFWwindow* createGLWindow()
{
	// glfw: initialize and configure
	// ------------------------------
	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

#ifdef __APPLE__
	glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif

	// glfw window creation
	// --------------------
	GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "AquaEvolution", NULL, NULL);
	if (window == NULL)
	{
		std::cout << "Failed to create GLFW window" << std::endl;
		glfwTerminate();
		exit(-1);
	}
	glfwMakeContextCurrent(window);
	glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
	//glfwSetKeyCallback(window, key_callback);

	// glad: load all OpenGL function pointers
	// ---------------------------------------
	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
	{
		std::cout << "Failed to initialize GLAD" << std::endl;
		exit(-1);
	}
	return window;
}
void createBuffers()
{
	createBackgroundBuffers();
	createAlgaBuffers();
	createFishBuffers();
}
void createBackgroundBuffers()
{
	// set up vertex data (and buffer(s)) and configure vertex attributes
	float vertices[] =
	{	// coords			// colors 
		 1.0f,  1.0f, 1.0f, SURFACE.r,		SURFACE.g,		SURFACE.b ,		// top right
		 1.0f, -1.0f, 0.0f, DEEPWATER.r,	DEEPWATER.g,	DEEPWATER.b ,	// bottom right
		-1.0f, -1.0f, 0.0f, DEEPWATER.r,	DEEPWATER.g,	DEEPWATER.b ,	// bottom left
		-1.0f,  1.0f, 0.0f, SURFACE.r,		SURFACE.g,		SURFACE.b ,		// top left 
	};
	unsigned int indices[] =
	{
		0, 1, 3,   // first triangle
		1, 2, 3    // second triangle
	};


	glGenVertexArrays(1, &VAO_bg);
	glGenBuffers(1, &VBO_bg);
	glGenBuffers(1, &EBO_bg);

	glBindVertexArray(VAO_bg);

	glBindBuffer(GL_ARRAY_BUFFER, VBO_bg);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO_bg);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

	// position attribute
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);
	// color attribute
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)(3 * sizeof(float)));
	glEnableVertexAttribArray(1);
}
void createAlgaBuffers()
{
	// set up vertex data (and buffer(s)) and configure vertex attributes
	float vertices[] =
	{	// coords			// colors 
		 0.0f,  1.0f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// top 
		 0.7f,  0.7f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// top right
		 1.0f,  0.0f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// left
		 0.7f, -0.7f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// bottom right 
		 0.0f, -1.0f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// bottom
		-0.7f, -0.7f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// bottom left 
		-1.0f,  0.0f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// left 
		-0.7f,  0.7f, 0.0f, ALGAECOLOR.r,	ALGAECOLOR.g,	ALGAECOLOR.b,	// top left 
	};
	unsigned int indices[] =
	{
		4, 5, 6,   
		4, 6, 7,   
		4, 7, 0,    
		4, 0, 1,    
		4, 1, 2,
		4, 2, 3
	};


	glGenVertexArrays(1, &VAO_alga);
	glGenBuffers(1, &VBO_alga);
	glGenBuffers(1, &EBO_alga);

	glBindVertexArray(VAO_alga);

	glBindBuffer(GL_ARRAY_BUFFER, VBO_alga);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO_alga);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

	// position attribute
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);
	// color attribute
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)(3 * sizeof(float)));
	glEnableVertexAttribArray(1);
}
void createFishBuffers()
{
	// set up vertex data (and buffer(s)) and configure vertex attributes
	float vertices[] =
	{	// coords			// colors 
		 0.0f,  1.0f, 0.0f, FISHCOLOR1.r,	FISHCOLOR1.g,	FISHCOLOR1.b,	// top 
		 0.3f,  0.0f, 0.0f, FISHCOLOR2.r,	FISHCOLOR2.g,	FISHCOLOR2.b,	// top right
		-0.3f,  0.0f, 0.0f, FISHCOLOR2.r,	FISHCOLOR2.g,	FISHCOLOR2.b,	// top left
		 0.0f, -0.5f, 0.0f, FISHCOLOR1.r,	FISHCOLOR1.g,	FISHCOLOR1.b,	// bottom 
		-0.4f, -1.0f, 0.0f, FISHCOLOR1.r,	FISHCOLOR1.g,	FISHCOLOR1.b,	// tail left
		 0.4f, -1.0f, 0.0f, FISHCOLOR1.r,	FISHCOLOR1.g,	FISHCOLOR1.b,	// tail right 
	};
	unsigned int indices[] =
	{
		2, 1, 0,
		2, 3, 1,
		4, 5, 3
	};


	glGenVertexArrays(1, &VAO_fish);
	glGenBuffers(1, &VBO_fish);
	glGenBuffers(1, &EBO_fish);

	glBindVertexArray(VAO_fish);

	glBindBuffer(GL_ARRAY_BUFFER, VBO_fish);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO_fish);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

	// position attribute
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);
	// color attribute
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)(3 * sizeof(float)));
	glEnableVertexAttribArray(1);
}

// --------------------------------------------- MEMORY MANAGEMENT FUNCTIONS ------------------------------------
void cleanup()
{
	// de-allocate all resources once they've outlived their purpose:
	glDeleteVertexArrays(1, &VAO_bg);
	glDeleteBuffers(1, &VBO_bg);
	glDeleteBuffers(1, &EBO_bg);

	glDeleteVertexArrays(1, &VAO_alga);
	glDeleteBuffers(1, &VBO_alga);
	glDeleteBuffers(1, &EBO_alga);

	glDeleteVertexArrays(1, &VAO_fish);
	glDeleteBuffers(1, &VBO_fish);
	glDeleteBuffers(1, &EBO_fish);

	// glfw: terminate, clearing all previously allocated GLFW resources.
	glfwTerminate();
}
// --------------------------------------------- RENDERING FUNCTIONS ----------------------------------------------
void renderLoop(GLFWwindow* window, Shader shader)
{
	// we operate in simple 2D space so form MVP matrices M is enough

	while (!glfwWindowShouldClose(window))
	{
		// allocate aquarium resources

		// input
		processInput(window);

		// render
		glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		// KERNEL HERE
		
		// model matrix
		glm::mat4 model = glm::mat4(1.0f);
		shader.use();

		// render background
		glBindVertexArray(VAO_bg);
		shader.setMat4("model", model);
		glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);

		// render algae
		glBindVertexArray(VAO_alga);
		model = glm::scale(model, glm::vec3(0.2, 0.2, 0.2));
		shader.setMat4("model", model);
		glDrawElements(GL_TRIANGLES, 18, GL_UNSIGNED_INT, 0);

		// render fish
		glBindVertexArray(VAO_fish);
		//model = glm::scale(model, glm::vec3(0.2, 0.2, 0.2));
		shader.setMat4("model", model);
		glDrawElements(GL_TRIANGLES, 9, GL_UNSIGNED_INT, 0);

		// glfw: swap buffers and poll IO events (keys pressed/released, mouse moved etc.)
		glfwSwapBuffers(window);
		glfwPollEvents();

	}
}

// --------------------------------------------- AQUARIUM MANAGEMENT FUNCTIONS ----------------------------------------------
void makeNewGeneration()
{
	// copy alive objects to new aquarium
	hostAquarium.readFromDeviceStruct(hostAquariumStruct, false);

	// make new objects from each one
	hostAquarium.newGeneration();

	// apply mutations and store objects in new arrays
	resetAquariumStruct(hostAquarium.objects.size());
	hostAquarium.writeToDeviceStruct(hostAquariumStruct);
}

// --------------------------------------------- MEMORY MANAGEMENT FUNCTIONS ----------------------------------------------
void resetAquariumStruct(int objectsCount)
{
	freeHostAquariumStruct();
	mallocHostAquariumStruct(objectsCount);
}
void copyAquariumStructToDevice()
{
	int n = *(hostAquariumStruct.objectsCount);
	checkCudaErrors(hipMemcpy(deviceAquariumStruct.objectsCount, hostAquariumStruct.objectsCount, sizeof(int), hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(deviceAquariumStruct.objects.positions.x, hostAquariumStruct.objects.positions.x, sizeof(float)*n, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(deviceAquariumStruct.objects.positions.y, hostAquariumStruct.objects.positions.y, sizeof(float)*n, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(deviceAquariumStruct.objects.directionVecs.x, hostAquariumStruct.objects.directionVecs.x, sizeof(float)*n, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(deviceAquariumStruct.objects.directionVecs.y, hostAquariumStruct.objects.directionVecs.y, sizeof(float)*n, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(deviceAquariumStruct.objects.alives, hostAquariumStruct.objects.alives, sizeof(bool)*n, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(deviceAquariumStruct.objects.fish, hostAquariumStruct.objects.fish, sizeof(bool)*n, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(deviceAquariumStruct.objects.sizes, hostAquariumStruct.objects.sizes, sizeof(float)*n, hipMemcpyHostToDevice));
}
void copyAquariumStructFromDevice()
{
	// NOTE: asuming sizes of arrays have not changed during kernel execution
	int n = *(hostAquariumStruct.objectsCount);
	
	checkCudaErrors(hipMemcpy(hostAquariumStruct.objectsCount, deviceAquariumStruct.objectsCount, sizeof(int), hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(hostAquariumStruct.objects.positions.x, deviceAquariumStruct.objects.positions.x, sizeof(float)*n, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(hostAquariumStruct.objects.positions.y, deviceAquariumStruct.objects.positions.y, sizeof(float)*n, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(hostAquariumStruct.objects.directionVecs.x, deviceAquariumStruct.objects.directionVecs.x, sizeof(float)*n, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(hostAquariumStruct.objects.directionVecs.y, deviceAquariumStruct.objects.directionVecs.y, sizeof(float)*n, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(hostAquariumStruct.objects.alives, deviceAquariumStruct.objects.alives, sizeof(bool)*n, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(hostAquariumStruct.objects.fish, deviceAquariumStruct.objects.fish, sizeof(bool)*n, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(hostAquariumStruct.objects.sizes, deviceAquariumStruct.objects.sizes, sizeof(float)*n, hipMemcpyHostToDevice));
}
void freeDeviceAquariumStruct()
{
	checkCudaErrors(hipFree(deviceAquariumStruct.objectsCount));

	checkCudaErrors(hipFree(deviceAquariumStruct.objects.positions.x));
	checkCudaErrors(hipFree(deviceAquariumStruct.objects.positions.y));

	checkCudaErrors(hipFree(deviceAquariumStruct.objects.directionVecs.x));
	checkCudaErrors(hipFree(deviceAquariumStruct.objects.directionVecs.y));

	checkCudaErrors(hipFree(deviceAquariumStruct.objects.alives));
	checkCudaErrors(hipFree(deviceAquariumStruct.objects.fish));
	checkCudaErrors(hipFree(deviceAquariumStruct.objects.sizes));
}
void freeHostAquariumStruct()
{
	free(hostAquariumStruct.objectsCount);

	free(hostAquariumStruct.objects.positions.x);
	free(hostAquariumStruct.objects.positions.y);

	free(hostAquariumStruct.objects.directionVecs.x);
	free(hostAquariumStruct.objects.directionVecs.y);

	free(hostAquariumStruct.objects.alives);
	free(hostAquariumStruct.objects.fish);
	free(hostAquariumStruct.objects.sizes);
}
void mallocHostAquariumStruct(int objectsCount)
{
	hostAquariumStruct.objectsCount = (int*)malloc(sizeof(int)); // -> may be unnecessary
	*(hostAquariumStruct.objectsCount) = objectsCount;
	hostAquariumStruct.objects.positions.x = (float*)malloc(objectsCount * sizeof(float));
	hostAquariumStruct.objects.positions.y = (float*)malloc(objectsCount * sizeof(float));
	hostAquariumStruct.objects.directionVecs.x = (float*)malloc(objectsCount * sizeof(float));
	hostAquariumStruct.objects.directionVecs.y = (float*)malloc(objectsCount * sizeof(float));
	hostAquariumStruct.objects.alives = (bool*)malloc(objectsCount * sizeof(bool));
	hostAquariumStruct.objects.fish = (bool*)malloc(objectsCount * sizeof(bool));
	hostAquariumStruct.objects.sizes = (float*)malloc(objectsCount * sizeof(float));
}
void mallocDeviceAquariumStruct(int objectsCount)
{
	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.objectsCount, sizeof(int)));

	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.objects.positions.x, sizeof(float)*objectsCount));
	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.objects.positions.y, sizeof(float)*objectsCount));

	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.objects.directionVecs.x, sizeof(float)*objectsCount));
	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.objects.directionVecs.y, sizeof(float)*objectsCount));

	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.objects.alives, sizeof(bool)*objectsCount));
	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.objects.fish, sizeof(bool)*objectsCount));
	checkCudaErrors(hipMalloc((void**)&deviceAquariumStruct.objects.sizes, sizeof(float)*objectsCount));
}